#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include <unordered_set>
#include <unordered_map>
#include <random>

__global__ void kernel(int *p){
    p[threadIdx.x] = 1;
}

__global__ void F1(float* y_true, float* y_pred, int m, int noClasses, float* y_trueEachClass, float* TP, float* FP, float* FN){
    for (int i = 0; i < noClasses; i++){
        if(y_pred[threadIdx.x] == y_true[threadIdx.x] && y_pred[threadIdx.x] == y_trueEachClass[i]) { atomicAdd(&TP[i], 1); printf("TP[%i]++\n", i); }
        if(y_pred[threadIdx.x] != y_true[threadIdx.x] && y_pred[threadIdx.x] == y_trueEachClass[i]) { atomicAdd(&FP[i], 1); printf("FP[%i]++\n", i); }
        if(y_pred[threadIdx.x] != y_true[threadIdx.x] && y_true[threadIdx.x] == y_trueEachClass[i]) { atomicAdd(&FP[i], 1); printf("FN[%i]++\n", i); }
    }
            
            
    
}


void getNoClasses(float* y_true, int m, int& noClasses){
    std::unordered_set<int> elementosUnicos;
    for (int i = 0; i < m; i++)
        elementosUnicos.insert(y_true[i]);             // Esta parte obtiene el No. clases
    noClasses = elementosUnicos.size();
}

void getVector(float* vector, int size){
    for (int i = 0; i < size; i++){ 
        if(i % 2 == 0) vector[i] = 1;
        else vector[i] = 2;
    }
        
        
}

void setClasses(float* y_true, int m, int& noClasses, float* y_trueEachClass){
    int temp = y_true[0];                                                 // Esta declaracion de temp nos servira para almacenar el valor de cada clase
    for (int i = 0; i < noClasses; i++)
    {
        y_trueEachClass[i] = temp;                                        // Con este algoritmo conseguimos llenar el array con longiud [no Classes]
        for (int e = 0; e < m; e++)                                       // con el valor dado de cada clase (ej. clase 0 = 45, clase 1 = 32)
           if (y_true[e] == y_trueEachClass[i]) y_true[e] = 0; 
        for (int e = 0; e < m; e++)
            if (y_true[e] != 0) { temp = y_true[e]; e = m; }    
    }
}

int main(){

    // float* y_pred, * y_true, * y_trueEachClass;
    float y_pred[6] = {0, 2, 1, 0, 0, 1}; float y_true [6] = {0, 1, 2, 0, 1, 2}; float* y_trueEachClass;

    float* TP, * FP, * FN;
    float* y_pred_d, * y_true_d, *y_trueEachClass_d;
    float* TP_d, * FP_d, * FN_d;
    int m = 6;                                                           // Dimensiones del array
    int noClasses = 0;


    // y_true = (float*)malloc(m * sizeof(float));                           // Reservacion de memoria del array de los valores objetivos y de los predecidos
    // y_pred = (float*)malloc(m * sizeof(float));
    // getVector(y_true, m);                                                 // Inicializacion de memoria de dicho array
    // getVector(y_pred, m);  

    hipMalloc((void**)&y_true_d, m * sizeof(float));
    hipMalloc((void**)&y_pred_d, m * sizeof(float));


    hipMemcpy(y_true_d, y_true, m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_pred_d, y_pred, m * sizeof(float), hipMemcpyHostToDevice);


    for (int i = 0; i < m; i++)
    {
        printf("[%f], ", y_true[i]);                                      // Para visualizar el array original
    }
    
    printf("\n\n");

    getNoClasses(y_true, m, noClasses);                                   // De esta funcion obtenemos: 1. No. clases, 2. vector con el valor dado de cada clase (ej. clase 0 = 45, clase 1 = 32)         
    printf("%i\n\n\n\n", noClasses);

    y_trueEachClass = (float*)malloc(noClasses * sizeof(float));          // Declaracion dinamica de un array con longitud [noClasses]
    TP = (float*)malloc(noClasses * sizeof(float));
    FP = (float*)malloc(noClasses * sizeof(float));
    FN = (float*)malloc(noClasses * sizeof(float));


    hipMalloc((void**)&y_trueEachClass_d, noClasses * sizeof(float));    // Alojacion en device dinamica del array con longitud [noClases]
    hipMalloc((void**)&TP_d, noClasses * sizeof(float));
    hipMalloc((void**)&FP_d, noClasses * sizeof(float));
    hipMalloc((void**)&FN_d, noClasses * sizeof(float));
    
    setClasses(y_true, m, noClasses, y_trueEachClass);                                                          // Obtenido el vector del tipo: clase 0 = 4, clase 1 = 9...
    hipMemcpy(y_trueEachClass_d, y_trueEachClass, noClasses * sizeof(float), hipMemcpyHostToDevice);          // Hacemos la transferencia de datos


    for (int i = 0; i < noClasses; i++)
    {
        printf("[%f], ", y_trueEachClass[i]);                             // Para visualizar el array nuevo
    }
    
    F1<<<1,m>>>(y_true_d, y_pred_d, m, noClasses, y_trueEachClass_d, TP_d, FP_d, FN);
    hipDeviceSynchronize();
    
    hipMemcpy(TP, TP_d, noClasses * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(FP, FP_d, noClasses * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(FN, FN_d, noClasses * sizeof(float), hipMemcpyDeviceToHost); 

    float F1 = 0;
    for (int i = 0; i < noClasses; i++)
        F1 += (TP[i] / (TP[i] + ((FP[i] + FN[i]) / 2)));
    F1 /= noClasses;
    
    printf("F1 MACRO: %f, TP_1: %f, FP_1: %f, FN_1: %f, TP_2: %f, FP_2: %f, FN_2: %f, TP_3: %f, FP_3: %f, FN_3: %f", F1, TP[0], FP[0], FN[0], TP[1], FP[1], FN[1], TP[2], FP[2], FN[2]);
    

    hipDeviceReset();

}