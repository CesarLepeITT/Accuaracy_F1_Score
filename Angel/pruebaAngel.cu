#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void kernel(float* mSemantica, float* targetValues, float* accuracyScore){
    if (mSemantica[((blockDim.y * blockIdx.y + threadIdx.y) * (gridDim.x * blockDim.x)) + (blockDim.x * blockIdx.x + threadIdx.x)] == targetValues[blockDim.x * blockIdx.x + threadIdx.x])
    {
        // printf("individuo: %i, thread y mSemantica[%i]: %f = target[%i]: %f\n", blockDim.y * blockIdx.y + threadIdx.y, ((blockDim.y * blockIdx.y + threadIdx.y) * (gridDim.x * blockDim.x)) + (blockDim.x * blockIdx.x + threadIdx.x), mSemantica[((blockDim.y * blockIdx.y + threadIdx.y) * (gridDim.x * blockDim.x)) + (blockDim.x * blockIdx.x + threadIdx.x)],blockDim.x * blockIdx.x +threadIdx.x, targetValues[blockDim.x * blockIdx.x + threadIdx.x]);
        atomicAdd(&accuracyScore[blockDim.y * blockIdx.y + threadIdx.y], 1);
    }
    __syncthreads();
    if (blockDim.x * blockIdx.x + threadIdx.x == m - 1){
        // printf("individuo: %i, accuracyscore: %f\n", blockDim.y * blockIdx.y + threadIdx.y, accuracyScore[blockDim.y * blockIdx.y + threadIdx.y]);
        
    }
}

__global__ void leer(float* mSemantica, float* targetValues, float* accuracyScore, int m){
    printf("thread: %i, y: %i\n",((blockDim.y * blockIdx.y + threadIdx.y) * (gridDim.x * blockDim.x)) + (blockDim.x * blockIdx.x + threadIdx.x),threadIdx.y);
     

}
void llenarVector(float* vector, int n, int m, int value){
    for (int i = 0; i < m; i++)
        vector[i] = 1;
}
void llenarMatriz(float* vector, int n, int m){
    for (int i = 0; i < n; i++){
        for (int e = 0; e < m; e++)
        {
            if (i % 2 == 0) { vector[i * m + e] = 1; }
            else {vector[i * m + e] = 0; }
        }
    }  
}



void splitM(int y, int x, int& newY, int& newX, int& gridY,  int& gridX){
    bool ok = true;
    bool ok2 = true;
    bool ok3 = true;
    int multiploY = 1;
    int multiploX = x;
    while (ok){
        if (x > pow (2, 31) - 1  && ok2){
            printf("hola");
            while (ok3){
                multiploX--;
                if ((x / multiploX) < pow (2, 31) - 1 && x % multiploX == 0) { ok = false; ok2 = false; }
            }
        }
        if (((y / multiploY) * (x / multiploX)) < 1024 && (y % multiploY) == 0 ) { ok = false; }
        else multiploY++;
    }
    newY = y / multiploY;
    newX = x / multiploX;
    gridY = (y + newY - 1) / newY;
    gridX = (x + newX - 1) / newX;

}


int main(){
    float* targetValues; float* mSemantica; float* accuracyScore;
    float* targetValues_d; float* mSemantica_d; float* accuracyScore_d;
    int n = 1024;
    int m = 1024;
    targetValues = (float*)malloc(m * sizeof(float));
    mSemantica = (float*)malloc(n * m * sizeof(float));
    accuracyScore = (float*)malloc(n * sizeof(float));

    llenarMatriz(mSemantica, n, m);
    llenarVector(targetValues, 1, m, 1);

    hipMalloc((void**)&targetValues_d, m * sizeof(float));
    hipMalloc((void**)&mSemantica_d, n * m * sizeof(float));
    hipMalloc((void**)&accuracyScore_d, n* sizeof(float));

    hipMemcpy(targetValues_d, targetValues, m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mSemantica_d, mSemantica, n * m * sizeof(float), hipMemcpyHostToDevice);

    int BdimY = 0;
    int BdimX = 0;

    int GdimY = 0;
    int GdimX = 0;


    splitM(n, m, BdimY, BdimX, GdimY, GdimX);
    dim3 block (BdimX,BdimY);
    dim3 grid(GdimX,GdimY);
    kernel<<<grid,block>>>(mSemantica_d, targetValues_d, accuracyScore_d, m);
    //leer<<<grid,block>>>(mSemantica_d, targetValues_d, accuracyScore_d, m);
    hipDeviceSynchronize();
    printf("%i, %i, %i\n",block.y, block.x, grid.x);
    hipMemcpy(accuracyScore, accuracyScore_d, n * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < n; i++)
    {
        printf("Accuracy Score [%i]: %f\n", i, accuracyScore[i] / m);
    }
    hipDeviceReset();
}