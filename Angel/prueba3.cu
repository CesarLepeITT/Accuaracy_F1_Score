#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// cada hilo evalua un individuo y por ende cada hilo ocupa acceder a los datos reales (true values)
// por ello, almacenaremos por bloque una porcion de los datos del array con los true values en la compartida, cada hilo (dentro del bloque)
// podra acceder a ella para hacer sus calculos, comparando la data correspondiente del individuo que le fue asignado
__global__ void accuracy_score(float *_true, float *y_pred, bool truE, int m, float *accuaracy)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; 
    // Operaciones de comparacion
    if (y_pred[m * threadIdx.y + threadIdx.x] == _true[ix])
    {
        atomicAdd(&accuaracy[threadIdx.y], 1);
        __syncthreads();
        printf("x: %i, y:%i score: %f \n", threadIdx.x, threadIdx.y, accuaracy[threadIdx.y]);
    }
    if (threadIdx.x == m - 1)
        accuaracy[threadIdx.y] /= m;
}

void FillingMatrices(float *matrix, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = 1;
}

void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        vector[i] = num;
    }
}

int main()
{
    // predicciones y valores esperados
    float *predictions;
    float *targValues;
    float *accuaracy;
    // float *dpredictions;
    // float *dtargValues;
    float *daccuaracy;

    // Matriz con 5 individuos y 6 columnas de datos (el array de valores esperados es de 6 elementos)
    int n = 1024;
    int m = 1024;
    int nm = n * m;

    // Sizes
    int sizeMatrix = nm * sizeof(float);
    int sizeVects = n * sizeof(float);

    // host
    accuaracy = (float *)malloc(sizeVects);
    hipHostMalloc((void **)&predictions, sizeMatrix, hipHostMallocDefault);
    hipHostMalloc((void **)&targValues, sizeVects, hipHostMallocDefault);

    // device
    hipMalloc((void **)&daccuaracy, sizeVects);

    // Inicializar matrices host
    FillingMatrices(predictions, n, m);
    Predictions(targValues, m, 1);
    Predictions(accuaracy, n, 0);

    // memcpy htd
    hipMemcpy(daccuaracy, accuaracy, sizeVects, hipMemcpyHostToDevice);
    // cudaMemcpy(dtargValues, targValues, sizeVects, cudaMemcpyHostToDevice);
    // cudaMemcpy(dpredictions, predictions, sizeMatrix, cudaMemcpyHostToDevice);

    dim3 block(m, n);
    accuracy_score<<<1, block>>>(targValues, predictions, false, m, daccuaracy);
    hipDeviceSynchronize();

    hipMemcpy(accuaracy, daccuaracy, sizeVects, hipMemcpyDeviceToHost);

    printf("[");
    for (int i = 0; i < n; i++)
    {
        if (i != n - 1)
            printf("%f, ", accuaracy[i]);
        else
            printf("%f", accuaracy[i]);
    }
    printf("]\n");

    // cudaFree(accuaracy);
    hipFree(targValues);
    hipFree(predictions);
    hipDeviceReset();
}