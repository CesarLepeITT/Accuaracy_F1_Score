#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// cada hilo evalua un individuo y por ende cada hilo ocupa acceder a los datos reales (true values)
// por ello, almacenaremos por bloque una porcion de los datos del array con los true values en la compartida, cada hilo (dentro del bloque)
// podra acceder a ella para hacer sus calculos, comparando la data correspondiente del individuo que le fue asignado
__global__ void accuracy_score(float *_true, float *y_pred, bool truE, int m, float *accuaracy)
{
    // Operaciones de comparacion
    if (y_pred[m * threadIdx.y + threadIdx.x] == _true[threadIdx.x])
    {
        atomicAdd(&accuaracy[threadIdx.y], 1);
        // printf("x: %i, y:%i score: %f \n", threadIdx.x, threadIdx.y, accuaracy[threadIdx.y]);
    }
    if(threadIdx.x == m - 1)
        accuaracy[threadIdx.y] /= m;
}

void FillingMatrices(float *matrix, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = 1;
}

void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        vector[i] = num;
    }
}

int main()
{
    // predicciones y valores esperados
    float *predictions;
    float *targValues;
    float *accuaracy;

    // Matriz con 5 individuos y 6 columnas de datos (el array de valores esperados es de 6 elementos)
    int n = 2;
    int m = 1024;
    int nm = n * m;

    // Sizes
    int sizeMatrix = nm * sizeof(float);
    int sizeVects = n * sizeof(float);

    //
    hipMallocManaged((void **)&predictions, sizeMatrix);
    hipMallocManaged((void **)&targValues, sizeVects);
    hipMallocManaged((void **)&accuaracy, sizeVects);

    // Inicializar matrices
    FillingMatrices(predictions, n, m);
    Predictions(targValues, m, 1);
    Predictions(accuaracy, n, 0);

    dim3 block(m, n);
    accuracy_score<<<1, block>>>(targValues, predictions, false, m, accuaracy);
    hipDeviceSynchronize();

    printf("[");
    for (int i = 0; i < n; i++)
    {
        if (i != n - 1)
            printf("%f, ", accuaracy[i]);
        else
            printf("%f", accuaracy[i]);
    }
    printf("]\n");

    hipDeviceReset();
}