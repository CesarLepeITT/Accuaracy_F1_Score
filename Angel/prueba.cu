#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


// cada hilo evalua un individuo y por ende cada hilo ocupa acceder a los datos reales (true values)
// por ello, almacenaremos por bloque una porcion de los datos del array con los true values en la compartida, cada hilo (dentro del bloque)
// podra acceder a ella para hacer sus calculos, comparando la data correspondiente del individuo que le fue asignado

__global__ void accuracy_score(int* _true, int* y_pred, bool truE, int m){
    __shared__ int trueS[16];
    if (m > 16)
    {
        int i = m / 16; // calculamos las veces que habra que almacenar espacios de 16 en la memoria compartida
        int e = m - i * 16; // calculamos lo que sobra de del multiplo anterior para al final hacer solo almacenar esa cantidad
        if (threadIdx.x < 16){
            for (int k = 0; k < i; k++){
                trueS [threadIdx.x] = _true[threadIdx.x + k * 16]; // los hilos necesarios copian los datos del vector de la data esperada (_true)
                if (threadIdx.x == 0){
                    printf("\n");
                    for (int j = 0; j < 16; j++)
                        printf ("%i, ", trueS[j]);
                }
            }
            if (threadIdx.x < e) {
                trueS [threadIdx.x] = _true[i * 16 + threadIdx.x];
                if (threadIdx.x == 0){
                    printf("\n");
                    for (int j = 0; j < e; j++)
                        printf ("%i, ", trueS[j]);
                }
            }
        }
            
    }
    
    __syncthreads();


}

void fillingMatrices(int* matrix, int n, int m){
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix [i * m + e] = e;
}

void predictions(int* vector, int m){
    for (int i = 0; i < m; i++)
    {
        vector[i] = i;
    }
    
}

int main(){
    // predicciones y valores esperados 
    int* predictions_h, * predictions_d, * targValues_h, * targValues_d;
    bool ban;

    // Matriz con 5 individuos y 6 columnas de datos (el array de valores esperados es de 6 elementos)
    int n = 5;
    int m = 33;



    size_t pitch;
    predictions_h = (int *)malloc(n * m * sizeof(int));
    targValues_h = (int*)malloc(m * sizeof(int)); // el tama;o del vector de valores esperados es del tama;o de columnas de datos
    fillingMatrices(predictions_h, n, m);
    //for (int i = 0; i < n; i++){
        //for (int e = 0; e < m; e++){
            //printf("|%i|", predictions_h[i * m + e]);
        //}
        //printf("\n");
    //}


    predictions(targValues_h, m);
    hipMallocPitch((void**)&predictions_d, &pitch, m * sizeof(int), n);
    hipMemcpy2D(predictions_d, pitch, predictions_h, m * sizeof(int), m * sizeof(int), n, hipMemcpyHostToDevice);
    hipMalloc((void**)&targValues_d, m * sizeof(int));
    hipMemcpy(targValues_d, targValues_h, m * sizeof(int), hipMemcpyHostToDevice);
    accuracy_score<<<1,32>>>(targValues_d, predictions_d, ban, m);

hipDeviceReset();
}