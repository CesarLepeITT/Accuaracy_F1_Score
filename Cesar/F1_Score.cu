#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

//TODO: Probar codigo con diferentes escenarios 
//TODO: Desarrollar mejor vesion en py del f1
//TODO: Limpiar codigo
//TODO: Optimizar el codigo

__global__ void F1_Score(float *y_true, float *y_pred, float *f1_score, int nx, int ny, unsigned int *aux)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int tid = iy * nx + ix;

    unsigned int cesar = iy * 2;
    unsigned int segunda = cesar + 1;
    unsigned int tercera = cesar + 2;
    if (tid < nx * ny && ix < nx && iy < ny)
    {
        // printf("ix%i iy%i tid %i cesar %i, segunda %i, tercera %i\n", ix, iy, tid, cesar, segunda, tercera);
        if (y_pred[tid] == 1 && y_true[ix] == 1) // TP A
        {
            atomicAdd(&aux[cesar], 1);
        }
        if (y_pred[tid] == 0 && y_true[ix] == 1) // FN B
        {
            atomicAdd(&aux[segunda], 1);
        }
        if (y_pred[tid] == 1 && y_true[ix] == 0) // FP C
        {
            atomicAdd(&aux[tercera], 1);
        }
        if (tid == nx - 1)
        {
            unsigned int a = aux[cesar];
            unsigned int b = aux[segunda];
            unsigned int c = aux[tercera];
            unsigned int x = (a + 0.5 * (b + c));
            if (x == 0){
                f1_score[iy] = -1;
                printf("Warning: Zero divition in f1_score[%i], value was set to -1.\n", iy);
            }                
            else
                f1_score[iy] = a / x;
        }
    }
}
void FillingMatrices(float *matrix, float num, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            matrix[i * m + j] = num;
}
void FillingMatrices(int *matrix, int num, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = num;
}
void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        if (i % 2 == 0)
            vector[i] = num;
        else
            vector[i] = 5;
    }
}
void VectorVacio(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        vector[i] = num;
    }
}
void PrintVect(float *vect, int ny)
{
    printf("[");
    for (int i = 0; i < ny; i++)
    {
        if (i != ny - 1)
            printf("%f, ", vect[i]);
        else
            printf("%f", vect[i]);
    }
    printf("]\n");
}

int main()
{
    // Set up dimensions
    int ny = 1;
    int nx = 8;
    int nm = ny * nx;

    // Memory size
    int nBytesPredictions = nm * sizeof(float);
    int nBytesTargetValues = nx * sizeof(float);
    int nBytesAccuracy = ny * sizeof(float);
    int nBytesAux = ny * 3 * sizeof(unsigned int);

    // Host memory allocation
    float *predictions, *targetValues, *h_accuracy;
    int *h_aux;
    h_accuracy = (float *)malloc(nBytesAccuracy);
    h_aux = (int *)malloc(nBytesAux);
    hipHostMalloc((void **)&predictions, nBytesPredictions, hipHostMallocDefault);
    hipHostMalloc((void **)&targetValues, nBytesTargetValues, hipHostMallocDefault);

    // Device memory allocation
    float *d_accuracy;
    unsigned int *d_aux;
    hipMalloc((void **)&d_accuracy, nBytesAccuracy);
    hipMalloc((void **)&d_aux, nBytesAux);

    // Host memory initialization
    // y_true = [1, 0, 1, 1, 0, 1, 0, 1]
    // y_pred = [1, 0, 1, 0, 1, 1, 0, 1]
    predictions[0] = 1;
    predictions[1] = 0;
    predictions[2] = 0;
    predictions[3] = 0;
    predictions[4] = 0;
    predictions[5] = 0;
    predictions[6] = 0;
    predictions[7] = 0;
    // FillingMatrices(predictions, 1, ny, nx);
    FillingMatrices(h_aux, 0, ny, 3);
    // Predictions(targetValues, nx, 1);
    targetValues[0] = 0;
    targetValues[1] = 0;
    targetValues[2] = 1;
    targetValues[3] = 0;
    targetValues[4] = 0;
    targetValues[5] = 0;
    targetValues[6] = 0;
    targetValues[7] = 1;
    VectorVacio(h_accuracy, nx, 0);

    // Memory transfer host to device
    hipMemcpy(d_accuracy, h_accuracy, nBytesAccuracy, hipMemcpyHostToDevice);
    hipMemcpy(d_aux, h_aux, nBytesAux, hipMemcpyHostToDevice);

    // Kernell call
    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    F1_Score<<<grid, block>>>(targetValues, predictions, d_accuracy, nx, ny, d_aux);
    hipDeviceSynchronize();

    // Memory transfer device to host
    hipMemcpy(h_accuracy, d_accuracy, nBytesAccuracy, hipMemcpyDeviceToHost);

    PrintVect(h_accuracy, ny);

    // Reset device
    hipDeviceReset();
}