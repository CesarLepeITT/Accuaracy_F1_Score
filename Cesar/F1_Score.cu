#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void f1_score(float *y_true, float *y_pred, float *f1_score, int nx, int ny, unsigned int *aux)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int tid = iy * nx + ix;
    printf("ix%i iy%i tid %i\n", ix, iy, tid);
    if (tid < nx * ny)
    {
        unsigned int cesar = 0;//iy * 2;
        unsigned int segunda = cesar + 1;
        unsigned int tercera = cesar + 2;
        if (y_pred[tid] == 1 && y_true[ix] == 1) // TP
        {
            atomicAdd(&aux[cesar], 1);
        }
        if (y_pred[tid] == 1 && y_true[ix] == 0) // FP
        {
            atomicAdd(&aux[segunda], 1);
        }
        if (y_pred[tid] == 0 && y_true[ix] == 1) // FN
        {
            atomicAdd(&aux[tercera], 1);
        }
        if (tid == nx - 1)
        {
            unsigned int a = aux[cesar];
            unsigned int b = aux[segunda];
            unsigned int c = aux[tercera];
            unsigned int x = (a + b) * (a + c);
            if(x == 0) 
                x = 1;
            float r = 2 * a / x;
            f1_score[iy] = r;
        }
    }
}
void FillingMatrices(float *matrix, float num,int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            matrix[i * m + j] = num;
}
void FillingMatrices(int *matrix, int num, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = num;
}
void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        if (i % 2 == 0)
            vector[i] = num;
        else
            vector[i] = 5;
    }
}
void VectorVacio(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        vector[i] = num;
    }
}
void PrintVect(float *vect, int ny)
{
    printf("[");
    for (int i = 0; i < ny; i++)
    {
        if (i != ny - 1)
            printf("%f, ", vect[i]);
        else
            printf("%f", vect[i]);
    }
    printf("]\n");
}

int main()
{
    // Set up dimensions
    int ny = 32;
    int nx = 2;
    int nm = ny * nx;

    // Memory size
    int nBytesPredictions = nm * sizeof(float);
    int nBytesTargetValues = nx * sizeof(float);
    int nBytesAccuracy = ny * sizeof(float);
    int nBytesAux = ny * 3 * sizeof(unsigned int);

    // Host memory allocation
    float *predictions, *targetValues, *h_accuracy;
    int *h_aux;
    h_accuracy = (float *)malloc(nBytesAccuracy);
    h_aux = (int *)malloc(nBytesAux);
    hipHostMalloc((void **)&predictions, nBytesPredictions, hipHostMallocDefault);
    hipHostMalloc((void **)&targetValues, nBytesTargetValues, hipHostMallocDefault);

    // Device memory allocation
    float *d_accuracy;
    unsigned int *d_aux;
    hipMalloc((void **)&d_accuracy, nBytesAccuracy);
    hipMalloc((void **)&d_aux, nBytesAux);

    // Host memory initialization
    FillingMatrices(predictions, 1, ny, nx);
    FillingMatrices(h_aux, 0, ny, 3);
    Predictions(targetValues, nx, 1);
    VectorVacio(h_accuracy, nx, 0);

    // Memory transfer host to device
    hipMemcpy(d_accuracy, h_accuracy, nBytesAccuracy, hipMemcpyHostToDevice);
    hipMemcpy(d_aux, h_aux, nBytesAux, hipMemcpyHostToDevice);


    // Kernell call
    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    f1_score<<<grid, block>>>(targetValues, predictions, d_accuracy, nx, ny, d_aux);
    hipDeviceSynchronize();

    // Memory transfer device to host
    hipMemcpy(h_accuracy, d_accuracy, nBytesAccuracy, hipMemcpyDeviceToHost);

    // PrintVect(h_accuracy, ny);

    // Reset device
    hipDeviceReset();
}