#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// cada hilo evalua un individuo y por ende cada hilo ocupa acceder a los datos reales (true values)
// por ello, almacenaremos por bloque una porcion de los datos del array con los true values en la compartida, cada hilo (dentro del bloque)
// podra acceder a ella para hacer sus calculos, comparando la data correspondiente del individuo que le fue asignado
__device__ float sumatoria;
__global__ void accuracy_score(float *_true, float *y_pred, bool truE, int nx, int ny, float *accuaracy)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; // 0 a 15
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // o 1 15
    unsigned int idx = iy * nx + ix;                         // Matriz
    unsigned int posx = idx - (iy * nx);

    // printf("ix: %i, iy:%i score: %i, idx%i \n", ix, iy, posx, idx);
    //  Operaciones de comparacion
    if (y_pred[idx] == _true[posx] && idx < nx * ny) // Corregir nx * nx a nx * ny
    {
        float sum = 1 / nx;
        atomicAdd(&accuaracy[iy], 1 / (float)nx); 
        //atomicAdd(reinterpret_cast<int*>(&accuaracy[iy]), __float_as_int(1 / nx)); 
        //accuaracy[iy] /= nx;      
        // printf("ix: %i, iy:%i posx: %i, idx%i, score: %f \n", ix, iy, posx, idx, accuaracy[iy]);
    }
   // if (posx == nx - 1)
   // {
    //    accuaracy[iy] /= nx;
   //     //printf("real score %f\n",accuaracy[iy]);
    //}
    __syncthreads();
}

void FillingMatrices(float *matrix, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = 1;
}

void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        if (i % 2 == 0)
            vector[i] = num;
        else 
            vector[i] = 5;
    }
}
void VectorVacio(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
            vector[i] = num;
    }
}


int main()
{
    // predicciones y valores esperados
    float *predictions;
    float *targValues;
    float *accuaracy;
    float *dpredictions;
    float *dtargValues;
    float *daccuaracy;

    // Matriz con 5 individuos y 6 columnas de datos (el array de valores esperados es de 6 elementos)
    int ny = 5;
    int nx = 5;
    int nm = ny * nx;

    // Sizes
    int sizePredictions = nm * sizeof(float);
    int sizeTargetValues = nx * sizeof(float);
    int sizeAccuracy = ny * sizeof(float);

    // host
    accuaracy = (float *)malloc(sizeAccuracy);
    targValues = (float *)malloc(sizeTargetValues);
    predictions = (float *)malloc(sizePredictions);

    // device
    hipMalloc((void **)&daccuaracy, sizeAccuracy);
    hipMalloc((void **)&dtargValues, sizeTargetValues);
    hipMalloc((void **)&dpredictions, sizePredictions);

    // Inicializar matrices host
    FillingMatrices(predictions, ny, nx);
    Predictions(targValues, nx, 1);
    VectorVacio(accuaracy, ny, 0);
    hipDeviceSynchronize();

    // memcpy htd
    hipMemcpy(daccuaracy, accuaracy, sizeAccuracy, hipMemcpyHostToDevice);
    hipMemcpy(dtargValues, targValues, sizeTargetValues, hipMemcpyHostToDevice);
    hipMemcpy(dpredictions, predictions, sizePredictions, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Kernell call
    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    accuracy_score<<<grid, block>>>(dtargValues, dpredictions, false, nx, ny, daccuaracy);
    hipDeviceSynchronize();

    hipMemcpy(accuaracy, daccuaracy, sizeAccuracy, hipMemcpyDeviceToHost);
    hipMemcpy(targValues, dtargValues, sizeTargetValues, hipMemcpyDeviceToHost);
    hipMemcpy(predictions, dpredictions, sizePredictions, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("[");
    for (int i = 0; i < ny; i++)
    {
        if (i != ny - 1)
            printf("%f, ", accuaracy[i]);
        else
            printf("%f", accuaracy[i]);
    }
    printf("]\n");

    hipDeviceReset();
}