#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// cada hilo evalua un individuo y por ende cada hilo ocupa acceder a los datos reales (true values)
// por ello, almacenaremos por bloque una porcion de los datos del array con los true values en la compartida, cada hilo (dentro del bloque)
// podra acceder a ella para hacer sus calculos, comparando la data correspondiente del individuo que le fue asignado
__device__ float sumatoria;
__global__ void accuracy_score(float *_true, float *y_pred, bool truE, int m, float *accuaracy)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Operaciones de comparacion
    if (y_pred[m * threadIdx.y + threadIdx.x] == _true[threadIdx.x])
    {   
        float sum = 1/m;
        atomicAdd(&accuaracy[threadIdx.y], sum);
        //printf("x: %i, y:%i score: %f ", threadIdx.x, threadIdx.y, accuaracy[threadIdx.y]);
        //printf(" %f \n",accuaracy[threadIdx.y]);
    }
    /*if (threadIdx.x == m - 1){
        accuaracy[threadIdx.y] /= m;
       // printf("a  %f",accuaracy[threadIdx.y]);
    }*/
    __syncthreads();
}

void FillingMatrices(float *matrix, int n, int m)
{
    for (int i = 0; i < n; i++)
        for (int e = 0; e < m; e++)
            matrix[i * m + e] = 1;
}

void Predictions(float *vector, int m, float num)
{
    for (int i = 0; i < m; i++)
    {
        vector[i] = num;
    }
}

int main()
{
    // predicciones y valores esperados
    float *predictions;
    float *targValues;
    float *accuaracy;
    float *dpredictions;
    float *dtargValues;
    float *daccuaracy;

    // Matriz con 5 individuos y 6 columnas de datos (el array de valores esperados es de 6 elementos)
    int ny = 128;
    int nx = 128;
    int nm = ny * nx;

    // Sizes
    int sizePredictions = nm * sizeof(float);
    int sizeTargetValues = nx * sizeof(float);
    int sizeAccuracy = ny * sizeof(float);

    // host
    accuaracy = (float *)malloc(sizeAccuracy);
    targValues = (float *)malloc(sizeTargetValues);
    predictions = (float *)malloc(sizePredictions);

    // device
    hipMalloc((void **)&daccuaracy, sizeAccuracy);
    hipMalloc((void **)&dtargValues, sizeTargetValues);
    hipMalloc((void **)&dpredictions, sizePredictions);

    // Inicializar matrices host
    FillingMatrices(predictions, ny, nx);
    Predictions(targValues, nx, 1);
    Predictions(accuaracy, ny, 0);
    hipDeviceSynchronize();

    // memcpy htd
    hipMemcpy(daccuaracy, accuaracy, sizeAccuracy, hipMemcpyHostToDevice);
    hipMemcpy(dtargValues, targValues, sizeTargetValues, hipMemcpyHostToDevice);
    hipMemcpy(dpredictions, predictions, sizePredictions, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Kernell call
    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    accuracy_score<<<grid, block>>>(dtargValues, dpredictions, false, nx, daccuaracy);
    hipDeviceSynchronize();

    hipMemcpy(accuaracy, daccuaracy, sizeAccuracy, hipMemcpyDeviceToHost);
    hipMemcpy(targValues, dtargValues, sizeTargetValues, hipMemcpyDeviceToHost);
    hipMemcpy(predictions, dpredictions, sizePredictions, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("[");
    for (int i = 0; i < ny; i++)
    {
        if (i != ny - 1)
            printf("%f, ", accuaracy[i]);
        else
            printf("%f", accuaracy[i]);
    }
    printf("]\n");

    hipDeviceReset();
}